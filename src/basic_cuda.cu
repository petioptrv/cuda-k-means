#include "hip/hip_runtime.h"
//
// Created by petioptrv on 2020-10-10.
//

#include <stdio.h>

#include "basic_cuda.h"
#include "helpers.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ double atomicAddc(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void assign_points_to_clusters(options_t *opts, int *n_vals,
        double *coords_d_points, int *cluster_d_points, double *min_dist_d_points,
        double *coord_d_centroids, double *prev_coord_d_centroids,
        double *n_points_d_centroids, double *sums_d_centroids) {
    int p = threadIdx.x + blockIdx.x * blockDim.x;
    int ops_count = opts->num_cluster * opts->dims;

    if (p < ops_count) {
        if (p < opts->num_cluster) {
            n_points_d_centroids[p] = 0;
        }
        sums_d_centroids[p] = 0;
        prev_coord_d_centroids[p] = coord_d_centroids[p];
    }

    if (p >= *n_vals) {
        return;
    }

    __syncthreads();

    min_dist_d_points[p] = __DBL_MAX__;

    long point_coord_offset = p * opts->dims;

    int cluster;
    for (int c = 0; c != opts->num_cluster; ++c) {
        double dist = 0;
        long centroid_coord_offset = c * opts->dims;
        for (int d = 0; d != opts->dims; d++) {

            // todo: unroll this further

            dist += (coord_d_centroids[centroid_coord_offset + d] - coords_d_points[point_coord_offset + d])
                    * (coord_d_centroids[centroid_coord_offset + d] - coords_d_points[point_coord_offset + d]);
        }

        if (dist < min_dist_d_points[p]) {
            min_dist_d_points[p] = dist;
            cluster_d_points[p] = c;
            cluster = c;
        }
    }

    atomicAddc(&n_points_d_centroids[cluster], 1);
    long centroid_coord_offset = cluster * opts->dims;
    for (int d = 0; d != opts->dims; ++d) {
        atomicAddc(&sums_d_centroids[centroid_coord_offset + d], coords_d_points[point_coord_offset + d]);
    }
}

__global__ void compute_new_centroids(options_t *opts,
        double *coord_d_centroids, double *prev_coord_d_centroids,
        double *n_points_d_centroids, double *sums_d_centroids, bool *converged_d_centroids) {
    int p = threadIdx.x + blockIdx.x * blockDim.x;
    int ops_count = opts->num_cluster * opts->dims;

    if (p >= ops_count) {
        return;
    }

    int c = p / opts->dims;

    coord_d_centroids[p] = sums_d_centroids[p] / n_points_d_centroids[c];

    __syncthreads();

    if (p % opts->dims == 0) {
        double dist = 0;
        for (int d = 0; d != opts->dims; d++) {

            // todo: unroll this further

            dist += (coord_d_centroids[p + d] - prev_coord_d_centroids[p + d])
                    * (coord_d_centroids[p + d] - prev_coord_d_centroids[p + d]);
        }
        dist = sqrt(dist);
        if (dist > opts->threshold) {
            converged_d_centroids[c] = false;
        } else {
            converged_d_centroids[c] = true;
        }
    }
}

void k_means_cuda_basic(point_t *points, centroid_t *centroids, options_t *opts, int *n_vals, timing_t *timing) {
    double *coords_d_points;
    int *cluster_d_points;
    double *min_dist_d_points;

    double *coord_d_centroids;
    double *prev_coord_d_centroids;
    double *n_points_d_centroids;
    double *sums_d_centroids;
    bool *converged_d_centroids;

    options_t *d_opts;
    int *d_n_vals;

    init_device_mem(points, centroids, opts, n_vals,
            &coords_d_points, &cluster_d_points, &min_dist_d_points,
            &coord_d_centroids, &prev_coord_d_centroids,
            &n_points_d_centroids, &sums_d_centroids, &converged_d_centroids,
            &d_opts, &d_n_vals);

    gpuErrchk( hipPeekAtLastError() )
    gpuErrchk( hipDeviceSynchronize() )

    int threads = opts->thread_count;
    int blocks;
    bool converged = false;

    struct timeval start, end;
    gettimeofday(&start, NULL);

    int e;
    for (e = 0; e != opts->max_num_iter or converged; ++e) {
        blocks = (*n_vals + threads - 1) / threads;
        assign_points_to_clusters<<<blocks, threads>>>(d_opts, d_n_vals,
                coords_d_points, cluster_d_points, min_dist_d_points,
                coord_d_centroids, prev_coord_d_centroids, n_points_d_centroids, sums_d_centroids);
        gpuErrchk( hipPeekAtLastError() )
        gpuErrchk( hipDeviceSynchronize() )

        blocks = (opts->num_cluster * opts->dims + threads - 1) / threads;
        compute_new_centroids<<<blocks, threads>>>(d_opts,
                coord_d_centroids, prev_coord_d_centroids, n_points_d_centroids, sums_d_centroids,
                converged_d_centroids);
        gpuErrchk( hipPeekAtLastError() )
        gpuErrchk( hipDeviceSynchronize() )

        copy_back_converged(centroids, opts, converged_d_centroids);
        gpuErrchk( hipPeekAtLastError() )
        gpuErrchk( hipDeviceSynchronize() )

        converged = true;
        for (int c = 0; c != opts->num_cluster; c++) {
            if (not centroids[c].converged) {
                converged = false;
                break;
            }
        }

        if (converged) break;
    }

    gettimeofday(&end, NULL);
    timing->total_iter_time = (end.tv_sec - start.tv_sec) * 1000 + (end.tv_usec - start.tv_usec) * .001;
    timing->iter_to_converge = e;

    copy_back_data(points, centroids, opts, n_vals, coords_d_points, cluster_d_points, min_dist_d_points,
            coord_d_centroids, prev_coord_d_centroids, n_points_d_centroids, sums_d_centroids, converged_d_centroids);
    gpuErrchk( hipPeekAtLastError() )
    gpuErrchk( hipDeviceSynchronize() )

    free_device_mem(coords_d_points, cluster_d_points, min_dist_d_points,
            coord_d_centroids, prev_coord_d_centroids, n_points_d_centroids, sums_d_centroids, d_opts, d_n_vals);
    gpuErrchk( hipPeekAtLastError() )
    gpuErrchk( hipDeviceSynchronize() )
}
